#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <vector>
#include <algorithm>
#include <numeric>

#include "Database.h"
#include "Query.h"
// #include "Execute.h"
#include "Execute_neworder.h"
#include "Execute_payment.h"
__global__ void KERNEL_CID(PAYMENT_SET *payment_set,
                           PAYMENT_QUERY *payment_query,
                           LOG *log,
                           SNAPSHOT *snapshot,
                           INDEX *index)
{ // get c_id if used c_name
    unsigned int thID = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int wID = thID / 32;
    for (size_t i = wID; i < PAYMENT_CNT; i += gridDim.x * blockDim.x / WARP_SIZE)
    {
        PAYMENT_NAMESPACE::get_c_id(i, payment_set, payment_query, log, snapshot, index);
    }
}
__global__ void KERNEL_EXECUTE(NEWORDER_SET *neworder_set,
                               PAYMENT_SET *payment_set,
                               NEWORDER_QUERY *neworder_query,
                               PAYMENT_QUERY *payment_query,
                               LOG *log,
                               SNAPSHOT *snapshot,
                               INDEX *index)
{ // execution phase
    unsigned int thID = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int wID = thID / 32;
    // while (true)
    // {
    unsigned int ID = wID / EXECUTE_WARP;
    if (wID % EXECUTE_WARP < 130)
    { // execute neworder_set
        // if (ID >= MINI_BATCH_CNT)
        // {
        //     return;
        // }
        NEWORDER_NAMESPACE::execute(ID * 32 + thID % 32, neworder_set, neworder_query, log, snapshot, index);
    }
    else
    { // execute payment_set
        // if (ID >= MINI_BATCH_CNT)
        // {
        //     return;
        // }
        PAYMENT_NAMESPACE::execute(ID * 32 + thID % 32, payment_set, payment_query, log, snapshot, index);
    }
    // }
}
__global__ void KERNEL_CHECK(NEWORDER_SET *neworder_set,
                             PAYMENT_SET *payment_set,
                             NEWORDER_QUERY *neworder_query,
                             PAYMENT_QUERY *payment_query,
                             LOG *log,
                             SNAPSHOT *snapshot,
                             INDEX *index)
{ // check conflicts phase
    unsigned int thID = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int wID = thID / 32;
    // while (true)
    // {
    unsigned int ID = wID / CHECK_WARP;
    if (wID % CHECK_WARP < 82)
    { // check neworder_set
        // if (ID >= MINI_BATCH_CNT)
        // {
        //     return;
        // }
        NEWORDER_NAMESPACE::check(ID * 32 + thID % 32, neworder_set, neworder_query, log, snapshot, index);
    }
    else
    { // check payment_set
        // if (ID >= MINI_BATCH_CNT)
        // {
        //     return;
        // }
        PAYMENT_NAMESPACE::check(ID * 32 + thID % 32, payment_set, payment_query, log, snapshot, index);
    }
    // }
}
__global__ void KERNEL_WRITEBACK(NEWORDER_SET *neworder_set,
                                 PAYMENT_SET *payment_set,
                                 NEWORDER_QUERY *neworder_query,
                                 PAYMENT_QUERY *payment_query,
                                 LOG *log,
                                 SNAPSHOT *snapshot,
                                 INDEX *index)
{ // writeback phase
    unsigned int thID = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int wID = thID / 32;
    // while (true)
    // {
    unsigned int ID = wID / WRITEBACK_WARP;
    if (wID % WRITEBACK_WARP < 63)
    { // writeback
        // if (ID >= MINI_BATCH_CNT)
        // {
        //     return;
        // }
        NEWORDER_NAMESPACE::write_back(ID * 32 + thID % 32, neworder_set, neworder_query, log, snapshot, index);
    }
    else
    { // writeback
        // if (ID >= MINI_BATCH_CNT)
        // {
        //     return;
        // }
        PAYMENT_NAMESPACE::write_back(ID * 32 + thID % 32, payment_set, payment_query, log, snapshot, index);
    }
    // }
    PAYMENT_NAMESPACE::reduce_update(thID, payment_set, payment_query, log, snapshot, index);
}

int main(int argc, char const *argv[])
{
    for (int cnt = 0; cnt < 1; cnt++)
    {
        hipSetDevice(SET_DEVICE);
        Database *database = new Database();
        initial_data(database);
        // database->copy_to_device();
        Query *query = new Query();
        initial_new_query(query);
        std::thread transfer_query;
        hipStream_t stream[STREAM_SIZE];
        for (size_t i = 0; i < STREAM_SIZE; i++)
        {
            hipStreamCreate(&stream[i]);
        }

        for (int epoch_ID = 0; epoch_ID < EPOCH_TP; epoch_ID++)
        {
            // make_new_query(epoch_ID, query);
            // hipStreamSynchronize();
            // query->random_choose_query(epoch_ID, stream);
            long long start_kernel_0_t = current_time();
            hipError_t err;
            int slotID = epoch_ID % SLOT_SIZE;
            query->random_choose_query(epoch_ID, stream);
            hipStreamSynchronize(stream[epoch_ID]);

            if (NEWORDER_PERCENT != 100)
            {
                // KERNEL_CID<<<GRID_SIZE, BLOCK_SIZE, 0>>>(query->payment_set_d + streamID,
                //                                          query->payment_query_d + streamID,
                //                                          database->log,
                //                                          database->snapshot_d,
                //                                          database->index_d);
                KERNEL_CID<<<GRID_SIZE, BLOCK_SIZE, 0, stream[epoch_ID]>>>(query->payment_set_d + slotID,
                                                                           query->payment_query_d + slotID,
                                                                           database->log,
                                                                           database->snapshot_d,
                                                                           database->index_d);
                err = hipGetLastError();
                if (err != hipSuccess)
                {
                    std::cout << "CUDA Error : " << hipGetErrorString(err) << std::endl;
                    break;
                }
                // hipDeviceSynchronize();
                hipStreamSynchronize(stream[epoch_ID]);
            }
            // KERNEL_EXECUTE<<<EXECUTE_GRID_SIZE, EXECUTE_BLOCK_SIZE, 0>>>(query->neworder_set_d + streamID,
            //                                                              query->payment_set_d + streamID,
            //                                                              query->neworder_query_d + streamID,
            //                                                              query->payment_query_d + streamID,
            //                                                              database->log,
            //                                                              database->snapshot_d,
            //                                                              database->index_d);
            KERNEL_EXECUTE<<<EXECUTE_GRID_SIZE, EXECUTE_BLOCK_SIZE, 0, stream[epoch_ID]>>>(query->neworder_set_d + slotID,
                                                                                           query->payment_set_d + slotID,
                                                                                           query->neworder_query_d + slotID,
                                                                                           query->payment_query_d + slotID,
                                                                                           database->log,
                                                                                           database->snapshot_d,
                                                                                           database->index_d);
            err = hipGetLastError();
            if (err != hipSuccess)
            {
                std::cout << "CUDA Error : " << hipGetErrorString(err) << std::endl;
                break;
            }
            // hipDeviceSynchronize();
            hipStreamSynchronize(stream[epoch_ID]);

            long long end_0 = current_time();
            // KERNEL_CHECK<<<CHECK_GRID_SIZE, CHECK_BLOCK_SIZE, 0>>>(query->neworder_set_d + streamID,
            //                                                        query->payment_set_d + streamID,
            //                                                        query->neworder_query_d + streamID,
            //                                                        query->payment_query_d + streamID,
            //                                                        database->log,
            //                                                        database->snapshot_d,
            //                                                        database->index_d);
            KERNEL_CHECK<<<CHECK_GRID_SIZE, CHECK_BLOCK_SIZE, 0, stream[epoch_ID]>>>(query->neworder_set_d + slotID,
                                                                                     query->payment_set_d + slotID,
                                                                                     query->neworder_query_d + slotID,
                                                                                     query->payment_query_d + slotID,
                                                                                     database->log,
                                                                                     database->snapshot_d,
                                                                                     database->index_d);
            err = hipGetLastError();
            if (err != hipSuccess)
            {
                std::cout << "CUDA Error : " << hipGetErrorString(err) << std::endl;
                break;
            }
            // hipDeviceSynchronize();
            hipStreamSynchronize(stream[epoch_ID]);
            long long end_1 = current_time();
            // KERNEL_WRITEBACK<<<WRITEBACK_GRID_SIZE, WRITEBACK_BLOCK_SIZE, 0>>>(query->neworder_set_d + streamID,
            //                                                                    query->payment_set_d + streamID,
            //                                                                    query->neworder_query_d + streamID,
            //                                                                    query->payment_query_d + streamID,
            //                                                                    database->log,
            //                                                                    database->snapshot_d,
            //                                                                    database->index_d);
            KERNEL_WRITEBACK<<<WRITEBACK_GRID_SIZE, WRITEBACK_BLOCK_SIZE, 0, stream[epoch_ID]>>>(query->neworder_set_d + slotID,
                                                                                                 query->payment_set_d + slotID,
                                                                                                 query->neworder_query_d + slotID,
                                                                                                 query->payment_query_d + slotID,
                                                                                                 database->log,
                                                                                                 database->snapshot_d,
                                                                                                 database->index_d);
            err = hipGetLastError();
            if (err != hipSuccess)
            {
                std::cout << "CUDA Error : " << hipGetErrorString(err) << std::endl;
                break;
            }
            // hipDeviceSynchronize();
            hipStreamSynchronize(stream[epoch_ID]);

            statistic_query(epoch_ID, query, stream);
            long long end_t = current_time();

            database->clear_LOG();

            float kernel_0_time = duration(start_kernel_0_t, end_t);
            float time_0 = duration(start_kernel_0_t, end_0);
            float time_1 = duration(end_0, end_1);
            float time_2 = duration(end_1, end_t);
            // std::cout << "Epoch " << epoch_ID << " kernel cost [" << kernel_0_time << " s].\n";
            // std::cout << "kernel 0 cost [" << time_0 << " s].\n";
            // std::cout << "kernel 1 cost [" << time_1 << " s].\n";
            // std::cout << "kernel 2 cost [" << time_2 << " s].\n";
            // check(database->log);
            if (epoch_ID >= WARMUP_TP && epoch_ID < EPOCH_TP - WARMUP_TP)
            {
                query->kernel_0_time_all += kernel_0_time;
                query->time_0_all += time_0;
                query->time_1_all += time_1;
                query->time_2_all += time_2;
            }

            hipDeviceSynchronize();
        }

        for (size_t i = 0; i < STREAM_SIZE; i++)
        {
            hipStreamDestroy(stream[i]);
        }

        // hipEventDestroy(start_event);
        // hipEventDestroy(end_event);
        // database->print();
        // query->print();
        float average_commit = ((float)query->commit_neworder + query->commit_payment) / (EPOCH_TP - 2 * WARMUP_TP);
        float average_n = ((float)query->commit_neworder) / (EPOCH_TP - 2 * WARMUP_TP);
        float average_p = ((float)query->commit_payment) / (EPOCH_TP - 2 * WARMUP_TP);
        float average_kernel_time = query->kernel_0_time_all / (EPOCH_TP - 2 * WARMUP_TP);
        float average_time_0 = query->time_0_all / (EPOCH_TP - 2 * WARMUP_TP);
        float average_time_1 = query->time_1_all / (EPOCH_TP - 2 * WARMUP_TP);
        float average_time_2 = query->time_2_all / (EPOCH_TP - 2 * WARMUP_TP);
        float average_copy_to_device = query->copy_to_device_cost / (EPOCH_TP - 2 * WARMUP_TP);
        float average_copy_to_host = query->copy_to_host_cost / (EPOCH_TP - 2 * WARMUP_TP);
        std::cout << "Kernel all cost [" << average_kernel_time << " s] in average.\n";
        std::cout << "Kernel 0 cost [" << average_time_0 << " s] in average.\n";
        std::cout << "Kernel 1 cost [" << average_time_1 << " s] in average.\n";
        std::cout << "Kernel 2 cost [" << average_time_2 << " s] in average.\n";
        std::cout << "copt to device cost [" << average_copy_to_device << " s] in average.\n";
        std::cout << "copy to host cost [" << average_copy_to_host << " s] in average.\n";
        std::cout << "memory copy speed is [" << query->memory_speed / (EPOCH_TP - 2 * WARMUP_TP) << " GB/s].\n";
        std::cout << "Commit [" << average_commit << "] in average.\n";
        std::cout << "Commit neworder [" << average_n << "] in average.\n";
        std::cout << "Commit payment [" << average_p << "] in average.\n";
        std::cout << "TPS [" << average_commit / average_kernel_time << "].\n";
        std::cout << "The warehouse size is [" << WAREHOUSE_SIZE << "].";
        std::cout << "The batch size is [" << BATCH_SIZE << "].\n\n";
        // std::cout << (WAREHOUSE_SIZE + DISTRICT_SIZE) * 2 * sizeof(unsigned int) * 31 << std::endl;
        free(database);
        free(query);
    }
    return 0;
}
